#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>

//GPU//GPU//GPU//GPU//GPU//GPU//GPU//GPU//GPU//GPU//GPU//GPU//GPU//GPU//GPU//GPU//GPU//GPU

#include <stdio.h>
#include <time.h>

#define SIZE	16

__global__ void Square(int* a, int* c, int n) //Добавляем __global__ чтобы функция выполнялась на GPU(многопоточно)
{
	int i = threadIdx.x; //Задаем как поточную переменную
	if (i < n) //Проверка наличия памяти для записи переменной
		c[i] = a[i] * a[i];
}

void SquareCPU(int* a, int* c, int n)
{
	int i;

	for (i = 0; i < n; ++i)
		c[i] = a[i] * a[i];
}

int main()
{
	float elapsedTime;
	clock_t start = clock();

	int* a, * c;	//Инициализация переменных
	int* d_a, * d_c;

	a = (int*)malloc(SIZE * sizeof(int));
	c = (int*)malloc(SIZE * sizeof(int));

	hipMalloc(&d_a, SIZE * sizeof(int)); //Выделение памяти на переменную
	hipMalloc(&d_c, SIZE * sizeof(int));

	for (int i = 0; i < SIZE; ++i)
	{
		a[i] = i;
		c[i] = 0;
	}

	hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice); //Копируем данные в GPU из CPU
	hipMemcpy(d_c, c, SIZE * sizeof(int), hipMemcpyHostToDevice);

	Square <<< 1, SIZE >> > (d_a, d_c, SIZE); //Вызов функции со специальной конфигурацией( <<< *кол-во блоков*, *величина блока*>>>...)

	hipMemcpy(c, d_c, SIZE * sizeof(int), hipMemcpyDeviceToHost); //Копируем данные обратно из GPU в CPU для вывода


	for (int i = 0; i < SIZE; ++i)
		printf("c[%d] = %d\n", i, c[i]);

	free(a);
	free(c);

	hipFree(d_a);
	hipFree(d_c); //Освобождаем память

	elapsedTime = ((double)clock() - start) / CLOCKS_PER_SEC; // Подсчет времени
	printf("GPU time elapsed: %f seconds \n", elapsedTime);

	//---------------------------------------------------------------------------------------------------------------------
	float elapsedTimeCPU;
	clock_t startCPU = clock();

	int* a1, * c1;
	a1 = (int*)malloc(SIZE * sizeof(int));
	c1 = (int*)malloc(SIZE * sizeof(int));

	for (int i = 0; i < SIZE; ++i)
	{
		a1[i] = i;
		c1[i] = 0;
	}

	SquareCPU(a1, c1, SIZE);

	for (int i = 0; i < SIZE; ++i)
		printf("c[%d] = %d\n", i, c1[i]);

	free(a1);
	free(c1);

	elapsedTimeCPU = ((double)clock() - startCPU) / CLOCKS_PER_SEC;
	printf("CPU time elapsed: %f seconds \n", elapsedTimeCPU);

	return 0;
}